#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cstdint>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <openssl/sha.h>

#define NUM_BLOCKS 256
#define THREADS_PER_BLOCK 256
#define TARGET_HASH "7461aafc4225e5d33b72e07e71abf5f8df35d7c8892e83fc2e44172607b48983"

__global__ void crackPasswordKernel(char* charset, int charsetLen, char* targetHash) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int charsetIdx[4] = {0, 0, 0, 0};
    char hash[SHA256_DIGEST_LENGTH];

    while (charsetIdx[0] < charsetLen) {
        // Generate a candidate password based on the current charset indices
        char candidate[5];
        candidate[0] = charset[charsetIdx[0]];
        candidate[1] = charset[charsetIdx[1]];
        candidate[2] = charset[charsetIdx[2]];
        candidate[3] = charset[charsetIdx[3]];
        candidate[4] = '\0';

        // Calculate the hash of the candidate password
        SHA256((const unsigned char*)candidate, strlen(candidate), (unsigned char*)hash);

        // Compare the generated hash with the target hash
        if (strcmp(targetHash, hash) == 0) {
            printf("Password found: %s\n", candidate);
            return;
        }

        // Increment charset indices
        charsetIdx[0]++;
        for (int i = 0; i < 4; i++) {
            if (charsetIdx[i] == charsetLen) {
                charsetIdx[i] = 0;
                charsetIdx[i + 1]++;
            }
        }
    }
}

int main() {
    char* charset = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
    int charsetLen = strlen(charset);

    // Target hash to crack (e.g., a SHA-256 hash)
    char targetHash[65]; // 64-character hash + null terminator
    strcpy(targetHash, TARGET_HASH);

    char* d_charset;
    char* d_targetHash;

    // Allocate memory on the GPU
    hipMalloc((void**)&d_charset, charsetLen);
    hipMalloc((void**)&d_targetHash, 65);

    // Copy data from host to GPU
    hipMemcpy(d_charset, charset, charsetLen, hipMemcpyHostToDevice);
    hipMemcpy(d_targetHash, targetHash, 65, hipMemcpyHostToDevice);

    // Launch the kernel
    crackPasswordKernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_charset, charsetLen, d_targetHash);
    hipDeviceSynchronize();

    // Free GPU memory
    hipFree(d_charset);
    hipFree(d_targetHash);

    return 0;
}
